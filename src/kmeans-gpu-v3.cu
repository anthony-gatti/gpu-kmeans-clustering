#include <iostream>
#include <cstdlib>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

struct Point {
    int id_cluster;
    double* values;
    std::string name;
};

struct Cluster {
    double* central_values;
};

__global__ void assignClusters(
    double *point_values,
    double *cluster_values,
    int *assignments,
    int total_points,
    int K,
    int total_values,
    int *changed_flag
) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= total_points) return;

    double min_dist = INFINITY;
    int best_cluster = -1;

    for (int c = 0; c < K; c++) {
        double sum = 0.0;
        for (int j = 0; j < total_values; j++) {
            double diff = cluster_values[c * total_values + j] - point_values[idx * total_values + j];
            sum += diff * diff;
        }
        if (sum < min_dist) {
            min_dist = sum;
            best_cluster = c;
        }
    }

    if (assignments[idx] != best_cluster) {
        assignments[idx] = best_cluster;
        atomicExch(changed_flag, 1);
    }
}

__global__ void updateCentroids(
    double *point_values,
    double *cluster_values,
    int *assignments,
    int *cluster_sizes,
    int total_points,
    int K,
    int total_values
) {
    int c = blockIdx.x;
    if (c >= K) return;

    extern __shared__ double shared_sums[];
    
    for (int j = threadIdx.x; j < total_values; j += blockDim.x) {
        shared_sums[j] = 0.0;
    }
    __syncthreads();

    int local_count = 0;
    for (int i = threadIdx.x; i < total_points; i += blockDim.x) {
        if (assignments[i] == c) {
            local_count++;
            for (int j = 0; j < total_values; j++) {
                atomicAdd(&shared_sums[j], point_values[i * total_values + j]);
            }
        }
    }

    atomicAdd(&cluster_sizes[c], local_count);
    __syncthreads();

    if (threadIdx.x == 0) {
        int count = cluster_sizes[c];
        if (count > 0) {
            for (int j = 0; j < total_values; j++) {
                cluster_values[c * total_values + j] = shared_sums[j] / count;
            }
        } else {
            for (int j = 0; j < total_values; j++) {
                cluster_values[c * total_values + j] = 0.0;
            }
        }
    }
}

long long kmeansCUDA(Point *h_points, Cluster *h_clusters, int total_points, int K, int total_values, int max_iterations) {
    auto begin = high_resolution_clock::now();

    double *d_point_values, *d_cluster_values;
    int *d_assignments, *d_cluster_sizes, *d_changed_flag;

    hipMalloc(&d_point_values, total_points * total_values * sizeof(double));
    hipMalloc(&d_cluster_values, K * total_values * sizeof(double));
    hipMalloc(&d_assignments, total_points * sizeof(int));
    hipMalloc(&d_cluster_sizes, K * sizeof(int));
    hipMalloc(&d_changed_flag, sizeof(int));

    // copies points into device memory
    for (int i = 0; i < total_points; i++) {
        hipMemcpy(d_point_values + i * total_values,
                              h_points[i].values,
                              total_values * sizeof(double),
                              hipMemcpyHostToDevice);
    }
    // copies initial centroids into device memory
    for (int i = 0; i < K; i++) {
        hipMemcpy(d_cluster_values + i * total_values,
                              h_clusters[i].central_values,
                              total_values * sizeof(double),
                              hipMemcpyHostToDevice);
    }
    // initialize assignments to -1.
    hipMemset(d_assignments, -1, total_points * sizeof(int));

    int threads = 256;
    int blocks_points = (total_points + threads - 1) / threads;

    auto end_phase1 = high_resolution_clock::now();

    int h_changed_flag = 0;
    int iter = 0;
    do {
        iter++;
        h_changed_flag = 0;
        hipMemset(d_changed_flag, 0, sizeof(int));

        assignClusters<<<blocks_points, threads>>>(d_point_values, d_cluster_values, d_assignments,
                                                     total_points, K, total_values, d_changed_flag);
        hipGetLastError();
        hipDeviceSynchronize();

        hipMemcpy(&h_changed_flag, d_changed_flag, sizeof(int), hipMemcpyDeviceToHost);

        hipMemset(d_cluster_sizes, 0, K * sizeof(int));

        updateCentroids<<<K, threads, total_values * sizeof(double)>>>(d_point_values, d_cluster_values,
                                                                         d_assignments, d_cluster_sizes,
                                                                         total_points, K, total_values);
        hipGetLastError();
        hipDeviceSynchronize();
    } while (h_changed_flag && iter < max_iterations);

    auto end = high_resolution_clock::now();

    int *assignments_host = new int[total_points];
    hipMemcpy(assignments_host, d_assignments, total_points * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < total_points; i++) {
        h_points[i].id_cluster = assignments_host[i];
    }
    delete[] assignments_host;

    for (int i = 0; i < K; i++) {
        hipMemcpy(h_clusters[i].central_values,
                              d_cluster_values + i * total_values,
                              total_values * sizeof(double),
                              hipMemcpyDeviceToHost);
    }

    long long duration = duration_cast<microseconds>(end - begin).count();

    // cout << "--------------------------------------------------" << endl;
    // for (int i = 0; i < K; i++) {
    //     cout << "Cluster " << i + 1 << endl;
    //     for (int j = 0; j < total_points; j++) {
    //         if (h_points[j].id_cluster == i) {
    //             cout << "Point " << j + 1 << ": ";
    //             for (int p = 0; p < total_values; p++) {
    //                 cout << h_points[j].values[p] << " ";
    //             }
    //             cout << endl;
    //         }
    //     }
    //     cout << "Cluster values: ";
    //     for (int j = 0; j < total_values; j++) {
    //         cout << h_clusters[i].central_values[j] << " ";
    //     }
    //     cout << "\n\n";
    // }
    // cout << "TOTAL EXECUTION TIME = " << duration << " microseconds" << endl;
    // cout << "TIME PHASE 1 = " << duration_cast<microseconds>(end_phase1 - begin).count() << " microseconds" << endl;
    // cout << "TIME PHASE 2 = " << duration_cast<microseconds>(end - end_phase1).count() << " microseconds" << endl;
    // cout << "--------------------------------------------------" << endl;

    hipFree(d_point_values);
    hipFree(d_cluster_values);
    hipFree(d_assignments);
    hipFree(d_cluster_sizes);
    hipFree(d_changed_flag);

    return duration;
}

int main(int argc, char *argv[]) {
    srand(10);

    int total_points, total_values, K, max_iterations, has_name;
    cin >> total_points >> total_values >> K >> max_iterations >> has_name;

    Point *points = new Point[total_points];
    for (int i = 0; i < total_points; i++) {
        points[i].values = new double[total_values];
        for (int j = 0; j < total_values; j++) {
            cin >> points[i].values[j];
        }
        if (has_name) {
            cin >> points[i].name;
        }
        points[i].id_cluster = -1;
    }

    cout << "K,AverageTimeMicroseconds" << endl;

    int k_vals[] = {2, 3, 5, 10, 20};
    int num_runs = 25;
    for (int k_val : k_vals) {
        long long total_time = 0;
        for (int r = 0; r < num_runs; r++) {
            Cluster *clusters = new Cluster[k_val];
            for (int i = 0; i < k_val; i++) {
                clusters[i].central_values = new double[total_values];
            }

            int *chosen = new int[k_val];
            for (int i = 0; i < k_val; i++) {
                while (true) {
                    int idx = rand() % total_points;
                    bool duplicate = false;
                    for (int j = 0; j < i; j++) {
                        if (chosen[j] == idx) { duplicate = true; break; }
                    }
                    if (!duplicate) {
                        chosen[i] = idx;
                        break;
                    }
                }
            }

            for (int i = 0; i < k_val; i++) {
                for (int j = 0; j < total_values; j++) {
                    clusters[i].central_values[j] = points[chosen[i]].values[j];
                }
            }
            delete[] chosen;

            long long run_time = kmeansCUDA(points, clusters, total_points, k_val, total_values, max_iterations);
            total_time += run_time;

            for (int i = 0; i < k_val; i++) {
                delete[] clusters[i].central_values;
            }
            delete[] clusters;
        }
        long long avg_time = total_time / num_runs;
        cout << k_val << "," << avg_time << endl;
    }

    for (int i = 0; i < total_points; i++) {
        delete[] points[i].values;
    }
    delete[] points;

    return 0;
}